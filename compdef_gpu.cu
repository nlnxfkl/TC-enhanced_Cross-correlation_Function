#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <mma.h>
#include <hip/hip_runtime.h>
#include "ptx_instructions.cuh"

// the number of template waves that are computed at once
#define N_BLOCK (NBLOCK * 2)

extern "C" {

constexpr int block_size = 96;
constexpr int warp_size = 32;
constexpr int num_warps = block_size / warp_size;

constexpr int m_mma = 16;
constexpr int n_mma = 8;
constexpr int k_mma = 8;

//FIXME: enable the computation for k > 256
constexpr int num_k_mmas = 32;
//const int num_mma_groups = (*nm - 1) / (num_mmas_per_group * k_mma) + 1;

constexpr int tpl_sh_size = (num_k_mmas * k_mma) * m_mma;
constexpr int obs_sh_size = (num_k_mmas * k_mma) + (num_warps * N_BLOCK * n_mma);
constexpr int result_sh_size = m_mma * (num_warps * N_BLOCK * n_mma);
constexpr int sh_size_in_bytes = 4 * (tpl_sh_size + obs_sh_size + result_sh_size);

__device__ __forceinline__ uint32_t _float_to_tf32(float f)
{
    uint32_t x = *reinterpret_cast<uint32_t*>(&f);
    if (isfinite(f))
    {
        x += 0x1000u;
    }
    return x;
}

__device__ __forceinline__ void _load_obs(
    uint32_t* obs_sh, const float* obs, const int nt, const int offset, const int tid)
{
    constexpr int num_elems = 16 / sizeof(int); // ensure 16B access
    float buffer[num_elems];
    uint32_t tmp[num_elems];
    for (int i_base = 0; i_base < obs_sh_size; i_base += block_size * num_elems)
    {
        const int i = i_base + tid * num_elems;
        if (i >= obs_sh_size)
        {
            break;
        }
        _global_load(buffer, &obs[offset + i]);
        #pragma unroll
        for (int j = 0; j < num_elems; j++)
        {
            tmp[j] = _float_to_tf32(buffer[j]);
        }
        _shared_store(&obs_sh[i], tmp);
    }
}

__device__ __forceinline__ int _swizzle_smem_pos_in(int x, int y)
{
    constexpr int vector_size = 4; // ensure 128bit access
    constexpr int num_vectors_per_row = (num_k_mmas * k_mma) / vector_size;
    const int new_x = ((x / vector_size) ^ (y % num_vectors_per_row)) * vector_size;
    return new_x;
}

__device__ __forceinline__ int _swizzle_smem_pos_out(int x, int y)
{
    constexpr int vector_size = 2; // ensure 64bit access
    constexpr int num_vectors_per_row = n_mma / vector_size;
    const int new_x = ((x / vector_size) ^ (y % num_vectors_per_row)) * vector_size;
    return new_x;
}

__device__ __forceinline__ void _load_tpl(
    float* tpl_sh, const float* tpl, const int nm, const int width, const int tid)
{
    constexpr int num_elems = 16 / sizeof(int);
    for (int i_base = 0; i_base < tpl_sh_size; i_base += block_size * num_elems)
    {
        const int i = i_base + tid * num_elems;
        if (i >= tpl_sh_size)
        {
            break;
        }
            
        const int gid_template = i / width;
        const int gid_time = i % width;
        const int gid = gid_time + nm * gid_template;

        const int sid_template = gid_template;
        const int sid_time = i % width;
        const int swizzled_sid_time = _swizzle_smem_pos_in(sid_time, sid_template);
        const int sid = swizzled_sid_time + width * sid_template;

        float buffer[num_elems];
        uint32_t tmp[num_elems];
        _global_load(buffer, &tpl[gid]);
        #pragma unroll
        for (int j = 0; j < num_elems; j++)
        {
            tmp[j] = _float_to_tf32(buffer[j]);
        }
        _shared_store(&tpl_sh[sid], tmp);
    }
}

__device__ __forceinline__ void _load_a_segment(
    uint32_t* av, float* tpl, const int nm, const int k_iter, const int k_mma, const int laneid)
{
    const int k_offset = k_iter * k_mma;

    const int row = laneid % 16;
    const int col = k_offset + (laneid / 16) * 4;
    const int swizzled_col = _swizzle_smem_pos_in(col, row);
    const int pos = swizzled_col + row * nm;

    _load_a_matrix(av, &tpl[pos]);
}

__device__ __forceinline__ void _load_b_segment(
    uint32_t* bv, const uint32_t* obs_sh, const int local_n_iter, const int n_mma, const int local_k_iter,
    const int k_mma, const int tid, const int laneid)
{
    const int n_iter = local_n_iter + (tid / warp_size) * N_BLOCK;
    const int col = laneid % 4;
    const int row = laneid / 4; 
    const int pos = (local_k_iter * k_mma) + (n_iter * n_mma) + col + row;

    bv[0] = obs_sh[pos];
    bv[1] = obs_sh[pos + 4];
}

__device__ __forceinline__ void store_results_to_smem(
    float* sh, float dv[NBLOCK][4], int warpid, int laneid)
{
    for (int local_n = 0; local_n < N_BLOCK; local_n++)
    {
        const int sh_offset = (warpid * N_BLOCK + local_n) * n_mma;

        const int col = (laneid % 4) * 2;

        const int row0 = laneid / 4;
        const int new_col0 = _swizzle_smem_pos_out(col, row0);
        const int sid0 = sh_offset + new_col0 + row0 * (n_mma * N_BLOCK * num_warps);

        const int row1 = row0 + 8;
        const int new_col1 = _swizzle_smem_pos_out(col, row1);
        const int sid1 = sh_offset + new_col1 + row1 * (n_mma * N_BLOCK * num_warps);

        *reinterpret_cast<float2*>(&sh[sid0]) = *reinterpret_cast<float2*>(&dv[local_n][0]);
        *reinterpret_cast<float2*>(&sh[sid1]) = *reinterpret_cast<float2*>(&dv[local_n][2]);
    } //local_n
}

__device__ __forceinline__ void store_results_to_gmem(
    float* co, float* sh, int nt, int tid)
{
    constexpr int num_elems = 8 / sizeof(int);
    for (int i_base = 0; i_base < m_mma * (n_mma * N_BLOCK * num_warps); i_base += block_size * num_elems)
    {
        const int i = i_base + tid * num_elems;
        const int width = n_mma * N_BLOCK * num_warps;
        const int gl_offset = blockIdx.x * width;
            
        const int gid_template = i / width;
        const int gid_time = gl_offset + i % width;
        if (gid_time >= nt) {
            continue;
        }
        const int gid = gid_time + nt * gid_template;

        const int sid_template = gid_template;
        const int sid_time = i % width;
        const int swizzled_sid_time = _swizzle_smem_pos_out((sid_time % 8), sid_template);
        const int sid = (sid_time / 8) * 8 + swizzled_sid_time + width * sid_template;

        *reinterpret_cast<float2*>(&co[gid]) = *reinterpret_cast<float2*>(&sh[sid]);
    }
}

__global__ void compdef_gputc_ (
    const int *nt_in, const int *nm_in, const int *nc_in, const float *obs, const float *tpl, float *co)
{
    const int nt = *nt_in;
    const int nm = *nm_in;
    const int nc = *nc_in;

    const int tid = threadIdx.x;
    const int laneid = tid % warp_size;
    const int warpid = tid / warp_size;

    extern __shared__ float sh[];
    float* tpl_sh = sh;
    uint32_t* obs_sh = reinterpret_cast<uint32_t*>(&tpl_sh[tpl_sh_size]);
    float* result_sh = reinterpret_cast<float*>(&obs_sh[obs_sh_size]);

    float dv[N_BLOCK][4] = {{ 0.0f }};

    // load all template wave components to shared memory.
    _load_tpl(tpl_sh, tpl, nm, (k_mma * num_k_mmas), tid);

    constexpr int n_per_block = num_warps * N_BLOCK * n_mma;
    #pragma unroll 1
    for (int obs_global_offset = blockIdx.x * n_per_block; obs_global_offset <= nc; obs_global_offset += gridDim.x * n_per_block)
    {
        // load all the observation wave to shared memory.
        _load_obs(obs_sh, obs, nt, obs_global_offset, tid);

        __syncthreads();

        uint32_t av[4], bv[2][2];

        #pragma unroll 16
        for (int k_mma_iter = 0; k_mma_iter < num_k_mmas; k_mma_iter++)
        {
            // load matrix A
            _load_a_segment(av, tpl_sh, (k_mma * num_k_mmas), k_mma_iter, k_mma, laneid);

            // load matrix B
            int b_reg_stage = 0;
            _load_b_segment(bv[b_reg_stage], obs_sh, 0, n_mma, k_mma_iter, k_mma, tid, laneid);

            #pragma unroll
            for (int n_mma_iter = 0; n_mma_iter < N_BLOCK; n_mma_iter++)
            { 
                // load matrix B
                if (n_mma_iter < N_BLOCK - 1)
                {
                    _load_b_segment(bv[b_reg_stage ^ 1], obs_sh, n_mma_iter + 1, n_mma, k_mma_iter, k_mma, tid, laneid);
                }

                // matrix-matrix product
                _tc_matmul(dv[n_mma_iter], av, bv[b_reg_stage], dv[n_mma_iter]);

                b_reg_stage ^= 1;

            } //n_mma_iter
        } //k_mma_iter

        store_results_to_smem(result_sh, dv, warpid, laneid);

        __syncthreads();

        store_results_to_gmem(co, result_sh, nt, tid);
    }
}

void call_compdef_gputc_ (
    const int *nt, const int *nm, const int *nc, const float *obs, const float *tpl, float *co)
{
    int device_id;
    hipGetDevice(&device_id);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    constexpr size_t dyn_sh_size = sh_size_in_bytes;
    hipFuncSetAttribute(reinterpret_cast<const void*>(compdef_gputc_), hipFuncAttributeMaxDynamicSharedMemorySize, dyn_sh_size);

    int num_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, compdef_gputc_, block_size, sh_size_in_bytes);
    int grid_size = num_blocks_per_sm * prop.multiProcessorCount;

    compdef_gputc_<<<grid_size, block_size, sh_size_in_bytes>>>
    (nt, nm, nc, obs, tpl, co);
}

}
